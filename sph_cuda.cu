#include "hip/hip_runtime.h"
/* smoothed particle hydrodynamics cuda routines */

/*
 *  (C) 2013 Janne Heikkarainen <janne.heikkarainen@tut.fi>
 *
 *  All rights reserved.
 *
 *  This file is part of Dubstep POSIX/CUDA Self-gravitating Smoothed Particle Hydrodynamics Simulator.
 *
 *  Dubstep is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  Dubstep is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with Dubstep.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "dubstep.h"
#include "sph_cuda.h"

#define BLOCK_SIZE 8
#define NUM_BLOCKS 8

__global__ void smoothing_length_iterator_kernel(float *r, float *origin, int *buffer, int *buffer_index, float *h, int n){
  // loop index
  int ii;

  // thread index
  int idx=blockIdx.x*blockDim.x+threadIdx.x;

  // number of particles in thread slice
  int slice_len=n/(BLOCK_SIZE*NUM_BLOCKS);

  // low and high bound of thread slice
  int lo=slice_len*idx;
  int hi=slice_len*idx+slice_len;

  // last thread handles remaining particles
  if(idx==(BLOCK_SIZE*NUM_BLOCKS)){
    hi=n;
  }

  // find particle neighbours in every thread slice
  for(ii=lo;ii<hi;ii++){
  }

  // wait for threads to finish
  __syncthreads();
}

void compute_smoothing_length_neighbours_cuda(struct universe *world, int iterations, int N_target){
  /* loop variables */
  int ii,jj,kk;

  /* state vector dimensions */
  int m;
  int n;

  /* pointers to state vectors */
  double *r_in;
  double *h_in;

  int *num_neighbours_in;

  /* host particle list buffer */
  int *buffer;

  /* device buffers */
  float *r_d;
  float *origin_d;

  int *buffer_d;
  int *buffer_index_d;

  float *h_d;
  float *h;

  int *buffer_index;

  float origin[3];

  // target for number of threads
  int num_threads=NUM_BLOCKS*BLOCK_SIZE;

  // compute execution configuration
  int blockSize=BLOCK_SIZE;
  int nBlocks=num_threads/blockSize;

  m=world->dim;  
  n=world->num;

  r_in=world->r2;
  h_in=world->h;

  num_neighbours_in=world->num_neighbours;

  // allocate particle list buffer on device
  hipMalloc((void**)&buffer_d, num_threads*n*sizeof(int));
  hipMalloc((void**)&buffer_index_d, num_threads*sizeof(int));

  // set up vector indeces
  buffer_index=(int*)malloc(num_threads*sizeof(int));
  for(ii=0;ii<num_threads;ii++)
    buffer_index[ii]=0;

  // allocate particle displacement vector on device
  hipMalloc((void**)&r_d, m*n*sizeof(float));

  // allocate particle origin vector on device
  hipMalloc((void**)&origin_d, m*sizeof(float));

  // copy particle displacement vector on device
  hipMemcpy(r_d, r_in, m*n*sizeof(float), hipMemcpyHostToDevice);

  // allocate smoothing length parameter on device
  hipMalloc((void**)&h_d, sizeof(float));

  // allocate smoothing length parameter on host
  h=(float*)malloc(sizeof(float));

  // allocate particle list buffer
  buffer=(int*)malloc(n*sizeof(int));

  /* iterate towards optimum number of neighbours */
  for(ii=0;ii<n;ii++){
    // particle origin displacement vector
    origin[0]=(float)r_in[m*ii+0];
    origin[1]=(float)r_in[m*ii+1];
    origin[2]=(float)r_in[m*ii+2];

    *h=(float)h_in[ii];
    
    // copy smoothing length parameter to device
    hipMemcpy(h_d, h, sizeof(float), hipMemcpyHostToDevice);
      
    // copy particle origin vector on device
    hipMemcpy(origin_d, &origin, m*sizeof(float), hipMemcpyHostToDevice);

    // copy empty buffer vector index on device
    hipMemcpy(buffer_index_d, buffer_index, num_threads*sizeof(int), hipMemcpyHostToDevice);

    if(world->neighbour_list[ii].list){
      world->neighbour_list[ii].num=0;
      free(world->neighbour_list[ii].list);
    }

    // call kernel
    smoothing_length_iterator_kernel <<< nBlocks, blockSize >>> (r_d, origin_d, buffer_d, buffer_index_d, h_d, n);

    // copy smoothing length parameter to host
    hipMemcpy(h, h_d, sizeof(float), hipMemcpyDeviceToHost);

    h_in[ii]=(double)*h;

    num_neighbours_in[ii]=num_threads;
    
    world->neighbour_list[ii].num=num_threads;
    world->neighbour_list[ii].list=(int*)malloc(num_threads*sizeof(int));
    if(!world->neighbour_list[ii].list){
      printf("Out of memory: particle neighbour list not allocated.\n");
      exit(1);
    }
    memcpy(world->neighbour_list[ii].list, buffer, num_threads*sizeof(int));
  }

  // clean up
  free(buffer);
  free(h);
  hipFree(h_d);
  hipFree(r_d);
  hipFree(origin_d);
  free(buffer_index);
  hipFree(buffer_index_d);
  hipFree(buffer_d);
}

