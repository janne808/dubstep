#include "hip/hip_runtime.h"
/* smoothed particle hydrodynamics cuda routines */

/*
 *  (C) 2013 Janne Heikkarainen <janne.heikkarainen@tut.fi>
 *
 *  All rights reserved.
 *
 *  This file is part of Dubstep POSIX/CUDA Self-gravitating Smoothed Particle Hydrodynamics Simulator.
 *
 *  Dubstep is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  Dubstep is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with Dubstep.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "dubstep.h"
#include "sph_cuda.h"

#define BLOCK_SIZE 8
#define NUM_BLOCKS 8

__global__ void smoothing_length_iterator_kernel(dubfloat_t *r, dubfloat_t *origin, int *buffer, int *buffer_index, dubfloat_t *h, int n){
  // loop index
  int ii;

  // thread index
  int idx=blockIdx.x*blockDim.x+threadIdx.x;

  // number of particles in thread slice
  int slice_len=n/(BLOCK_SIZE*NUM_BLOCKS);

  // low and high bound of thread slice
  int lo=slice_len*idx;
  int hi=slice_len*idx+slice_len;

  // last thread handles remaining particles
  if(idx==(BLOCK_SIZE*NUM_BLOCKS)){
    hi=n;
  }

  // find particle neighbours in every thread slice
  //for(ii=lo;ii<hi;ii++){
  //}

  // wait for threads to finish
  __syncthreads();
}

void compute_smoothing_length_neighbours_cuda(struct universe *world, int iterations, int N_target){
  /* loop variables */
  int ii;
  int jj;
  int kk;

  /* state vector dimensions */
  int m;
  int n;

  /* pointers to state vectors */
  dubfloat_t *r_in;
  dubfloat_t *h_in;

  /* device buffers */
  dubfloat_t *r_d;

  /* pointer to particle neighbour number vector */
  int *num_neighbours_in;

  /* maximum list length */
  int max_list_len;

  /* flattened neighbour list on device mem */
  int *flat_list_d;

  /* allocate particle displacement vector on device */
  hipMalloc(&r_d, 3*world->num*sizeof(dubfloat_t));

  /* copy particle displacement vector on device */
  hipMemcpy(r_d, world->r, 3*world->num*sizeof(dubfloat_t), hipMemcpyHostToDevice);    

  /* flatten out neighbour list */
  /* search for max list size */
  max_list_len=0;
  for(ii=0;ii<world->num;ii++){
	if(world->neighbour_list[ii].max_size>max_list_len)
		max_list_len=world->neighbour_list[ii].max_size;
  }

  /* add slack to max size for new neighbours */
  max_list_len+=50;

  /* set up flat list to device memory */
  /* allocate list on device */
  hipMalloc(&flat_list_d, world->num*max_list_len*sizeof(int));

  /* free device list */
  hipFree(flat_list_d);

  /* free device particle displacement vector */
  hipFree(r_d);
}

